#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ReDi.h"
#include <math.h>

/****

     tu[] is u input grid
     u[] is u output grid
     tv[] is v input grid
     v[] is v output grid
     w is width of grid

     compute 4-nearest neighbor updates 

****/

__global__ void cuUpdateGridGlobal(double *u, double *tu, double *v, double *tv, int w) {
  
  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
  double u1, v1, uv2;

  // printf("(%i, %i)", i, j);

  uv2 = dataAt(tu, i, j, w) * dataAt(tv, i, j, w) * dataAt(tv, i, j, w);

  // printf("yay");

  u1 = dataAt(tu, i, j, w)
    + 0.2 * (dataAt(tu, i+1, j, w) + dataAt(tu, i-1, j, w)
            + dataAt(tu, i, j+1, w) + dataAt(tu, i, j-1, w)
            - 4 * dataAt(tu, i, j, w))
    - uv2 + 0.025 * (1 - dataAt(tu, i, j, w));

  u1 = fmin(1, u1);
  dataAt(u, i, j, w) = fmax(0, u1);

  
  v1 = dataAt(tv, i, j, w)
    + 0.1 * (dataAt(tv, i+1, j, w) + dataAt(tv, i-1, j, w)
            + dataAt(tv, i, j+1, w) + dataAt(tv, i, j-1, w)
            - 4 * dataAt(tv, i, j, w))
    + uv2 - 0.08 * dataAt(tv, i, j, w);
  v1 = fmin(1, v1);
  dataAt(v, i, j, w) = fmax(0, v1);

  // printf("(%f, %f)", fmax(0, u1), fmax(0, v1));
}

__global__ void cuUpdateGridShared(double *u, double *tu, double *v, double *tv, int w) {
  
  int i = blockIdx.y * blockDim.y;
  int j = blockIdx.x * blockDim.x;
  double u1, v1, uv2;
  __shared__ double su[TILEWIDTH+2][TILEWIDTH+2];
  __shared__ double sv[TILEWIDTH+2][TILEWIDTH+2];

  int linId = threadIdx.y * blockDim.x + threadIdx.x;

  int si = linId / (TILEWIDTH+2);
  int sj = linId % (TILEWIDTH+2);

  su[si][sj] = dataAt(tu, (i+si), (j+sj), w);
  sv[si][sj] = dataAt(tv, (i+si), (j+sj), w);

  // printf("copy ij (%i, %i) to sij (%i, %i) ", i+si, j+sj, si, sj);
  // printf("\n");

  linId += blockDim.x * blockDim.y;

  si = linId / (TILEWIDTH+2);
  sj = linId % (TILEWIDTH+2);

  if (si < TILEWIDTH+2 && sj < TILEWIDTH+2) {
    su[si][sj] = dataAt(tu, (i+si), (j+sj), w);
    sv[si][sj] = dataAt(tv, (i+si), (j+sj), w);
  }

  // printf("copy ij (%i, %i) to sij (%i, %i) ", i+si, j+sj, si, sj);
  // printf("\n");

  __syncthreads();

  // printf("(%i, %i)", i, j);

  i += threadIdx.y + 1;
  j += threadIdx.x + 1;

  si = threadIdx.y + 1;
  sj = threadIdx.x + 1;


  // if (su[si][sj] != dataAt(tu, i, j, w)){
  //   printf("incorrect u copy at ij (%i, %i) %lf sij (%i, %i) %lf ", i, j, dataAt(tu, i, j, w), si, sj, su[si][sj]);
  // }
  // if (sv[si][sj] != dataAt(tv, i, j, w)){
  //   printf("incorrect v copy at ij (%i, %i) %lf sij (%i, %i) %lf ", i, j, dataAt(tv, i, j, w), si, sj, sv[si][sj]);
  // }

  uv2 = su[si][sj] * sv[si][sj] * sv[si][sj];

  // printf("yay");

  u1 = su[si][sj]
    + 0.2 * (su[si+1][sj] + su[si-1][sj]
            + su[si][sj+1] + su[si][sj-1]
            - 4 * su[si][sj])
    - uv2 + 0.025 * (1 - su[si][sj]);

  u1 = fmin(1, u1);
  dataAt(u, i, j, w) = fmax(0, u1);

  
  v1 = sv[si][sj]
    + 0.1 * (sv[si+1][sj] + sv[si-1][sj]
            + sv[si][sj+1] + sv[si][sj-1]
            - 4 * sv[si][sj])
    + uv2 - 0.08 * sv[si][sj];
  v1 = fmin(1, v1);
  dataAt(v, i, j, w) = fmax(0, v1);

  // printf("(%f, %f)", fmax(0, u1), fmax(0, v1));
}

/*
void initGrid(double u0[], double u1[], double v0[], double v1[], int w) {
  int i, j;

  for (i=0; i<w; i++) {
    for (j=0; j<w; j++) {
      dataAt(u0, i, j, w) = 1.0;
      dataAt(u1, i, j, w) = 1.0;
      dataAt(v0, i, j, w) = 0.0;
      dataAt(v1, i, j, w) = 0.0;
    }
  }
}
*/

